#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <cmath>
#include <algorithm>
#include <vector>
#include <chrono>
#include <type_traits>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>

class GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

public:
    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

template <class Resolution = std::chrono::microseconds>
class CPUTimer {
public:
    using Clock = std::conditional_t<std::chrono::high_resolution_clock::is_steady,
        std::chrono::high_resolution_clock,
        std::chrono::steady_clock>;

private:
    const Clock::time_point mStart = Clock::now();

public:
    CPUTimer() = default;
    ~CPUTimer() {
        const auto end = Clock::now();
        std::ostringstream strStream;
        // strStream << "Destructor Elapsed: "
        //     << std::chrono::duration_cast<Resolution>(end - mStart).count()
        //     << std::endl;
        // std::cout << strStream.str() << std::endl;
    }

    void stop(std::string const& label) {
        const auto end = Clock::now();
        std::ostringstream strStream;
        strStream << "Stop Elapsed " << label << " : "
            << std::chrono::duration_cast<Resolution>(end - mStart).count()
            << std::endl;
        std::cout << strStream.str() << std::endl;
    }
};

constexpr unsigned NUM_BINS = 256;

void histogramCPU(unsigned int* input, unsigned int* res, unsigned int n)
{
    for (unsigned i=0; i<n; ++i) {
        res[input[i]]++;
    }
}

__global__ void histogramGPUGlobalMem(unsigned *input, unsigned *res, unsigned n)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (size_t i = tid; i < n; i += (gridDim.x * blockDim.x)) {
        atomicAdd(&res[input[i]], 1);
    }
}

__global__ void histogramGPUSharedMem(unsigned *input, unsigned *res, unsigned n)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Allocate a local histogram for each TB
    __shared__ int s_res[NUM_BINS];

    // Initalize the shared memory to 0
    if (threadIdx.x < NUM_BINS) {
        s_res[threadIdx.x] = 0;
    }

    // Wait for shared memory writes to complete
    __syncthreads();

    for (size_t i = tid; i < n; i += (gridDim.x * blockDim.x)) {
        atomicAdd(&s_res[input[i]], 1);
    }

    // Wait for shared memory writes to complete
    __syncthreads();

    // Combine the partial results
    if (threadIdx.x < NUM_BINS) {
        atomicAdd(&res[threadIdx.x], s_res[threadIdx.x]);
    }
}

int main()
{
    unsigned N = 1 << 16;

    std::vector<unsigned> h_input(N);
    std::vector<unsigned> h_result(NUM_BINS);
    thrust::host_vector<unsigned> h_result_copy(NUM_BINS);

    std::srand(42);
    std::generate(std::begin(h_input), std::end(h_input), [] { return  (unsigned)rand() % NUM_BINS; } );

    CPUTimer<> cpu_timer;
    histogramCPU(h_input.data(), h_result.data(), h_result.size());
    cpu_timer.stop("histogramCPU");

    thrust::device_vector<unsigned> d_input(h_input);
    thrust::device_vector<unsigned> d_result(h_result_copy);
    thrust::device_vector<unsigned> d_result2(h_result_copy);

    // Number of threads per threadblock
    size_t THREADS = 512;

    // Calculate the number of threadblocks
    size_t BLOCKS = N / THREADS;

    GpuTimer timer;
    timer.Start();
    CPUTimer<> cpu_timer1;
    histogramGPUGlobalMem<<<BLOCKS, THREADS>>>(thrust::raw_pointer_cast(d_input.data()),
                                               thrust::raw_pointer_cast(d_result.data()),
                                               d_result.size());
    timer.Stop();
    cpu_timer1.stop("histogramGPUGlobalMem");

    std::cout << "elapsed for histogramGPUGlobalMem: " << timer.Elapsed() << " millisecs\n";

    timer.Start();
    CPUTimer<> cpu_timer2;
    histogramGPUSharedMem<<<BLOCKS, THREADS>>>(thrust::raw_pointer_cast(d_input.data()),
                                               thrust::raw_pointer_cast(d_result2.data()),
                                               d_result2.size());
    timer.Stop();
    cpu_timer2.stop("histogramGPUGlobalMem");
    std::cout << "elapsed for histogramGPUSharedMem: " << timer.Elapsed() << " millisecs\n";

    auto compare_vals = [](std::string const& label,
                           auto const& arr1,
                           auto const& arr2) {
           std::cout << "checking values: " << label << std::endl;
           size_t mismatches = 0;
           for (size_t i = 0; i < arr1.size(); ++i) {
               if (arr1[i] != arr2[i]) {
                   ++mismatches;                                   
                   std::cout << "mismatch for index: " << i << " got:" << arr1[i] << " expected: " << arr2[i] << std::endl;
               }
           }
           std::cout << mismatches << " mismatches for " << label << std::endl;

    };

    // copy back
    h_result_copy = d_result;

    compare_vals("h_result vs. d_result", h_result, h_result_copy);
    // copy back
    h_result_copy = d_result2;

    compare_vals("h_result vs. d_result2", h_result, h_result_copy);

}
