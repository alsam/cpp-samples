#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <string>

#define THREADBLOCK_SIZE 128
#define WORKING_SET_SIZE_ELEM_BITS 21
#define WORKING_SET_SIZE_ELEMS (1 << WORKING_SET_SIZE_ELEM_BITS)
#define ITERATION_COUNT 5

// FNV-1a released into public domain
#define INITIAL_HASH 14695981039346656037ULL
__forceinline__ __device__ unsigned long long hash(unsigned long long current_hash, int val)
{
   return (current_hash ^ (unsigned long long)val) * 1099511628211ULL;
}

template<bool write>
__launch_bounds__(THREADBLOCK_SIZE) 
__global__ void
access_random(float * in_buf, int coalescing_mask)
{
	int elem_id = blockIdx.x * THREADBLOCK_SIZE + threadIdx.x;
	int base_read_elem_id = elem_id & coalescing_mask;
	int thread_local_id = base_read_elem_id - elem_id;

	float res = 0.0F;
	unsigned long long current_hash = INITIAL_HASH;
	for(int i = 0; i < ITERATION_COUNT; ++i)
	{
		current_hash = hash(current_hash, base_read_elem_id + i);
		int offset = (current_hash + thread_local_id * 8) & ((1 << WORKING_SET_SIZE_ELEM_BITS) - 1);
		res += __ldg(in_buf + offset);
		if (write)
			in_buf[offset] = 1.0F;
	}
	if (elem_id == res)
		in_buf[elem_id & ((1 << WORKING_SET_SIZE_ELEM_BITS) - 1)] = res;
}

int main(int argc, char *argv[])
{
	float * d_buffer_in;

	int coalescing_mask = ~0;
	int threadblock_count = 65536 * 4;
	std::string mode = "read";

	if (argc > 1)
		coalescing_mask = ~(atol(argv[1]) - 1);
	if (argc > 2)
		mode = argv[2];
	if (argc > 3)
		threadblock_count = atol(argv[3]);

	std::cout << "coalescing mask = " << coalescing_mask << ", threadblock_count = " << threadblock_count
		<< ", mode = " << mode << std::endl;
	hipMalloc((void **)&d_buffer_in, sizeof(float) * WORKING_SET_SIZE_ELEMS);

	if (mode == "read")
		access_random<false><<<threadblock_count, THREADBLOCK_SIZE>>>(d_buffer_in, coalescing_mask);
	else if (mode == "write")
		access_random<true><<<threadblock_count, THREADBLOCK_SIZE>>>(d_buffer_in, coalescing_mask);
	hipDeviceSynchronize();

	hipFree(d_buffer_in);

	hipDeviceReset();

	return 0;
}

